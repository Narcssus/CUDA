
#include "hip/hip_runtime.h"


#include <stdio.h>

hipError_t VectorSum(int *c, const int *a, const int *b, unsigned int size);

__global__ void VectorSum(int *c, const int *a, const int *b)
{
	int i = threadIdx.x;
	c[i] = a[i] + b[i];
}

int main()
{
	const int arraySize = 5;
	const int a[arraySize] = { 1, 2, 3, 4,5};
	const int b[arraySize] = { 10, 20, 30, 40, 50 };
	int c[arraySize] = { 0 };
	int d=1;
	hipError_t cudaStatus = VectorSum(c, a, b, arraySize);
	if (cudaStatus != hipSuccess) {
		printf( "addWithCuda failed!");
		return 1;
	}
	printf("\n{1,2,3,4,5} + {10,20,30,40,50} = {%d,%d,%d,%d,%d}\n",
		c[0], c[1], c[2], c[3], c[4]);
	cudaStatus = hipDeviceReset();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceReset failed!");
		return 1;
	}
	getchar();

	return 0;
}
hipError_t VectorSum(int *c, const int *a, const int *b, unsigned int size)
{
	int *dev_a = 0;
	int *dev_b = 0;
	int *dev_c = 0;
	hipError_t err = hipGetLastError();
	if(err!=hipSuccess){
		fprintf(stderr,hipGetErrorString(err));
		
	}
	hipMalloc((void**)&dev_a, size * sizeof(int));
	err = hipGetLastError();
	if(err!=hipSuccess){
		fprintf(stderr,hipGetErrorString(err));
		
	}

	hipMalloc((void**)&dev_b, size * sizeof(int));
	err = hipGetLastError();
	if(err!=hipSuccess){
		fprintf(stderr,hipGetErrorString(err));
		
	}
	hipMalloc((void**)&dev_c, size * sizeof(int));
	err = hipGetLastError();
	if(err!=hipSuccess){
		fprintf(stderr,hipGetErrorString(err));
		
	}
	hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
	err = hipGetLastError();
	if(err!=hipSuccess){
		fprintf(stderr,hipGetErrorString(err));
		
	}
	hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
	err = hipGetLastError();
	//printf("adsdsadasdsa");
	//fprintf(stderr,cudaGetErrorString(err));
	if(err!=hipSuccess){
		fprintf(stderr,hipGetErrorString(err));
		
	}

	hipMemcpy(dev_c, c, size * sizeof(int), hipMemcpyHostToDevice);
	err = hipGetLastError();
	if(err!=hipSuccess){
		fprintf(stderr,hipGetErrorString(err));
		
	}
	VectorSum<<<1, size>>>(c, dev_a, dev_b);
	err = hipGetLastError();
	if (err != hipSuccess) {
		fprintf(stderr, hipGetErrorString(err));

	}
	err = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
	if (err != hipSuccess) {
		fprintf(stderr, hipGetErrorString(err));
	}
	return err;
}
