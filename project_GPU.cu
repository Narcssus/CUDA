#include "hip/hip_runtime.h"
#include <iostream> 
#include <stdio.h>
#include <stdlib.h> 
#include "hip/hip_runtime.h"
#include ""
#include <stdlib.h>
#include <stdio.h>
#include <math.h>
#include <Windows.h>
#include "hip/device_functions.h"
#include <hip/hip_runtime.h>
using namespace std; 
#define villageSize 6 //��ׯ��С
#define random(x) (rand()%x)

void NextGeneration_CPU(int **village,int **next_village)
{
	int villagers=0;
	for (int i=1;i<=villageSize;i++){
		for(int j=1;j<=villageSize;j++){
			villagers=0;
			/*����ÿ��������Χ�Ĵ������*/
			villagers=village[i-1][j-1]	+village[i-1][j]+village[i-1][j+1]
			+village[i][j-1]	+0				+village[i][j+1]
			+village[i+1][j-1]+village[i+1][j]+village[i+1][j+1];
			if(villagers==3)			//�������������һ������
				*(*(next_village+i)+j)=1;
			else if(villagers==2)		//�������������һ��������һ��״̬
				*(*(next_village+i)+j)=*(*(village+i)+j);
			else						//���������һ��������
				*(*(next_village+i)+j)=0;
		}
	}
}



__global__ void NextGeneration_GPU(int *g_odata, int *g_idata)
{
	int j=threadIdx.x+threadIdx.y*blockDim.x;
	int sum=blockDim.x;
	int villagers[1024];
	if(j%(villageSize+2)==0||j%(villageSize+2)==villageSize+1||j<villageSize+2||j>=(villageSize+2)*(villageSize+1))	villagers[j]=0;
	else
	{
		villagers[j]=0;
		/*����ÿ��������Χ�Ĵ������*/
		villagers[j]=	*(g_idata+j-villageSize-3)
			+*(g_idata+j-villageSize-2)
			+*(g_idata+j-villageSize-1)
			+*(g_idata+j-1)
			+*(g_idata+j+1)
			+*(g_idata+j+villageSize+1)
			+*(g_idata+j+villageSize+2)
			+*(g_idata+j+villageSize+3);

		if(villagers[j]!=2&&villagers[j]!=3)	//�����Ϊ2��3������һ������
			*(g_odata+j)=0;
		if(villagers[j]==3)				//�������������һ������
			*(g_odata+j)=1;
		if(villagers[j]==2)				//�������������һ��������һ��״̬
			*(g_odata+j)=*(g_idata+j);
		int a=0;
	}
}





int main() 
{
	int Generation=1;	//����
	/*��ʱ����*/
	LARGE_INTEGER t_start,t_end,freq;
	float ms;
	QueryPerformanceFrequency(&freq);
	//GPU �����ʼ��
	int *GPUvillageA, *GPUvillageB;//ʹ���������齻���ʾ��������
	GPUvillageA = (int *)malloc(sizeof(int *) * (villageSize+2)* (villageSize+2));
	GPUvillageB = (int *)malloc(sizeof(int *) * (villageSize+2)* (villageSize+2));
	for(int i=0;i<=villageSize+1;i++){
		for(int j=0;j<=villageSize+1;j++){
			GPUvillageA[i*(villageSize+2)+j]=0;
			GPUvillageB[i*(villageSize+2)+j]=0;
		}
	}
	/*ʹ����������������,40%�Ĵ����*/
	for(int i=1;i<=villageSize;i++){
		for(int j=1;j<=villageSize;j++){
			if(random(10)>4)
				GPUvillageA[i*(villageSize+2)+j]=1;
			else
				GPUvillageA[i*(villageSize+2)+j]=0;
		}
	}

	//CPU �����ʼ��
	int **CPUvillageA, **CPUvillageB;//ʹ���������齻���ʾ��������
	/*������̬�ռ�*/
	CPUvillageA = (int **)malloc(sizeof(int *) * villageSize+2);
	CPUvillageB = (int **)malloc(sizeof(int *) * villageSize+2);
	for(int i=0; i<villageSize+2; i++)
	{
		CPUvillageA[i] = (int *)malloc(sizeof(int) * villageSize+2);
		CPUvillageB[i] = (int *)malloc(sizeof(int) * villageSize+2);
	}
	/*��ʼ������*/
	for(int i=0;i<=villageSize+1;i++){
		*((CPUvillageA[i])+0)=0;
		*((CPUvillageA[0])+i)=0;
		*((CPUvillageA[i])+villageSize+1)=0;
		*((CPUvillageA[villageSize+1])+i)=0;
	}
	/*ʹ����������������,40%�Ĵ����*/
	for(int i=1;i<=villageSize;i++){
		for(int j=1;j<=villageSize;j++){
			if(random(10)>4)
				*((CPUvillageA[i])+j)=1;
			else
				*((CPUvillageA[i])+j)=0;
		}
	}


	/*CPU����*/
	QueryPerformanceCounter(&t_start);
	for(Generation=1;Generation<1000;Generation++)
	{
		if(Generation%2==1){
			NextGeneration_CPU(CPUvillageA,CPUvillageB);
		}
		else{
			NextGeneration_CPU(CPUvillageB,CPUvillageA);
		}
	}
	QueryPerformanceCounter(&t_end);
	ms=1e3*(t_end.QuadPart-t_start.QuadPart)/freq.QuadPart;
	cout<<"CPU runTime: "<<ms<<endl;

	/*GPU����*/
	int num_elements, mem_size;
	int *d_idata, *d_odata;	

	num_elements = (villageSize+2)*(villageSize+2);
	mem_size     = sizeof(int) * num_elements;


	hipMalloc((void**)&d_idata, mem_size);
	hipMalloc((void**)&d_odata, mem_size);

	hipMemcpy(d_idata, GPUvillageA, mem_size, hipMemcpyHostToDevice);
	hipMemcpy(d_odata, GPUvillageB, mem_size, hipMemcpyHostToDevice);


	QueryPerformanceCounter(&t_start);

	for(Generation=1;Generation<1000;Generation++)
	{
		if(Generation%2==1){
			if(num_elements<1024)
				NextGeneration_GPU<<<1,num_elements>>>(d_odata,d_idata);
			else
				NextGeneration_GPU<<<num_elements/1024,1024>>>(d_odata,d_idata);
			hipMemcpy(GPUvillageB, d_odata, mem_size,hipMemcpyDeviceToHost);			
			hipMemcpy(d_idata, GPUvillageB, mem_size, hipMemcpyHostToDevice);
		}
		else{
			if(num_elements<1024)
				NextGeneration_GPU<<<1,num_elements>>>(d_odata,d_idata);
			else
				NextGeneration_GPU<<<num_elements/1024,1024>>>(d_odata,d_idata);
			hipMemcpy(GPUvillageA, d_odata, mem_size,hipMemcpyDeviceToHost);						
			hipMemcpy(d_idata, GPUvillageA, mem_size, hipMemcpyHostToDevice);
		}
	}

	QueryPerformanceCounter(&t_end);
	ms=1e3*(t_end.QuadPart-t_start.QuadPart)/freq.QuadPart;
	cout<<"GPU runTime: "<<ms<<endl;



	free(GPUvillageA);
	free(GPUvillageB);

	hipFree(d_idata);
	hipFree(d_odata);
	getchar();
	hipDeviceReset();
	return 0;

}
