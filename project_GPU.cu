#include "hip/hip_runtime.h"
#include <iostream> 
#include <stdio.h>
#include <stdlib.h> 
#include "hip/hip_runtime.h"
#include ""
#include <stdlib.h>
#include <stdio.h>
#include <math.h>
#include <Windows.h>
#include "hip/device_functions.h"
#include <hip/hip_runtime.h>
using namespace std; 
#define villagerSize 10000
#define random(x) (rand()%x)

void NextGeneration_CPU(int **village,int **next_village)
{
	int villagers=0;
	for (int i=1;i<=villagerSize;i++){
		for(int j=1;j<=villagerSize;j++){
			villagers=0;
			villagers=village[i-1][j-1]	+village[i-1][j]+village[i-1][j+1]
			+village[i][j-1]	+0				+village[i][j+1]
			+village[i+1][j-1]+village[i+1][j]+village[i+1][j+1];
			if(villagers==3)
				*(*(next_village+i)+j)=1;
			else if(villagers==2)
				*(*(next_village+i)+j)=*(*(village+i)+j);
			else
				*(*(next_village+i)+j)=0;
		}
	}
}



__global__ void NextGeneration_GPU(int *g_odata, int *g_idata)
{
	int j = threadIdx.x;
	int sum=blockDim.x;
	int villagers[1024];
	if(j%(villagerSize+2)==0||j%(villagerSize+2)==villagerSize+1||j<villagerSize+2||j>=(villagerSize+2)*(villagerSize+1))	villagers[j]=0;
	else
	{
		villagers[j]=0;
		villagers[j]=	*(g_idata+j-villagerSize-3)
			+*(g_idata+j-villagerSize-2)
			+*(g_idata+j-villagerSize-1)
			+*(g_idata+j-1)
			+*(g_idata+j+1)
			+*(g_idata+j+villagerSize+1)
			+*(g_idata+j+villagerSize+2)
			+*(g_idata+j+villagerSize+3);

		if(villagers[j]!=2&&villagers[j]!=3)
			*(g_odata+j)=0;
		if(villagers[j]==3)
			*(g_odata+j)=1;
		if(villagers[j]==2)
			*(g_odata+j)=*(g_idata+j);
		int a=0;
	}
}





int main() 
{
	int Generation=1;
	 LARGE_INTEGER t_start,t_end,freq;
  float ms;
  QueryPerformanceFrequency(&freq);
	int *villageA, *villageB;
	villageA = (int *)malloc(sizeof(int *) * (villagerSize+2)* (villagerSize+2));
	villageB = (int *)malloc(sizeof(int *) * (villagerSize+2)* (villagerSize+2));
	for(int i=0;i<=villagerSize+1;i++){
		for(int j=0;j<=villagerSize+1;j++){
			villageA[i*(villagerSize+2)+j]=0;
			villageB[i*(villagerSize+2)+j]=0;
		}
	}

	for(int i=1;i<=villagerSize;i++){
		for(int j=1;j<=villagerSize;j++){
			if(random(10)>4)
				villageA[i*(villagerSize+2)+j]=1;
			else
				villageA[i*(villagerSize+2)+j]=0;
		}
	}
	for(int i=1;i<=villagerSize;i++){
		for(int j=1;j<=villagerSize;j++){
	//		cout<<villageA[i*(villagerSize+2)+j]<<" ";
		}
	//	cout<<endl;
	}
//	cout<<"OK"<<endl;
//	getchar();

	//CPU
	/*
	for(Generation=1;Generation<3000;Generation++)
	{
	if(Generation%2==1){
	NextGeneration_CPU(villageA,villageB);
	for(int i=1;i<=villagerSize;i++){
	for(int j=1;j<=villagerSize;j++){
	cout<<villageB[i][j]<<" ";
	}
	cout<<endl;
	}
	}
	else{
	NextGeneration_CPU(villageB,villageA);
	for(int i=1;i<=villagerSize;i++){
	for(int j=1;j<=villagerSize;j++){
	cout<<villageA[i][j]<<" ";
	}
	cout<<endl;
	}
	}
	getchar();
	}
	*/







	int num_elements, num_threads, mem_size, shared_mem_size;

	int *d_idata, *d_odata;


	num_elements = (villagerSize+2)*(villagerSize+2);
	num_threads  = num_elements;
	mem_size     = sizeof(int) * num_elements;


	hipMalloc((void**)&d_idata, mem_size);
	hipMalloc((void**)&d_odata, mem_size);

	hipMemcpy(d_idata, villageA, mem_size, hipMemcpyHostToDevice);
	hipMemcpy(d_odata, villageB, mem_size, hipMemcpyHostToDevice);

	shared_mem_size = sizeof(int) * num_elements;


	  QueryPerformanceCounter(&t_start);

	for(Generation=1;Generation<10;Generation++)
	{
		if(Generation%2==1){
			NextGeneration_GPU<<<1,num_threads>>>(d_odata,d_idata);
			hipMemcpy(villageB, d_odata, mem_size,hipMemcpyDeviceToHost);			
				for(int i=1;i<=villagerSize;i++){
					for(int j=1;j<=villagerSize;j++){
				//		cout<<villageB[i*(villagerSize+2)+j]<<" ";
					}
				//	cout<<endl;
				}
			hipMemcpy(d_idata, villageB, mem_size, hipMemcpyHostToDevice);
		}
		else{
			NextGeneration_GPU<<<1,num_threads>>>(d_odata,d_idata);
			hipMemcpy(villageA, d_odata, mem_size,hipMemcpyDeviceToHost);						
				for(int i=1;i<=villagerSize;i++){
					for(int j=1;j<=villagerSize;j++){
				//		cout<<villageA[i*(villagerSize+2)+j]<<" ";
					}
				//	cout<<endl;
				}
			hipMemcpy(d_idata, villageA, mem_size, hipMemcpyHostToDevice);
		}
	//	cout<<endl;
		 

	//	cout<<"finished"<<endl;
	//	getchar();
	}

	 QueryPerformanceCounter(&t_end);
	   ms=1e3*(t_end.QuadPart-t_start.QuadPart)/freq.QuadPart;
cout<<"times"<<ms<<endl;



		free(villageA);
		free(villageB);

		hipFree(d_idata);
		hipFree(d_odata);
		getchar();
		hipDeviceReset();
		return 0;

	}
