#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <hip/hip_runtime.h>
int main()
{
	hipError_t status;
	int numDevices;
	hipGetDeviceCount(&numDevices);
	printf("%d devices detected:\n",numDevices);
	hipDeviceProp_t  device;

	for(int i=0;i<numDevices;i++)
	{
		char szName[256];
		hipGetDeviceProperties(&device,i);
		hipDeviceProp_t sDevProp = device;
		printf( "�豸����: %s\n", sDevProp.name );
		printf( "����������:%d���μ�������: %d\n", sDevProp.major,sDevProp.minor );
		printf( "�豸����ȫ���ڴ�: %0.lf\n",(double) sDevProp.totalGlobalMem);
		printf( "ÿ�߳̿�����߳���: %d\n", sDevProp.maxThreadsPerBlock);
		printf( "�豸����ȫ���ڴ�����: %d\n", sDevProp.totalConstMem);
		printf( "ÿ�߳̿���ù����ڴ�����: %d\n", sDevProp.sharedMemPerBlock );
		printf( "ÿ�߳̿���üĴ�������: %d\n", sDevProp.regsPerBlock );
		printf( "�豸�еĴ�����������: %d\n", sDevProp.multiProcessorCount);
		printf( "ÿ�������������פ���߳���: %d\n", sDevProp.maxThreadsPerMultiProcessor );
		printf( "�߳������߳�����: %d\n", sDevProp.warpSize );
		printf( "ȫ���ڴ����߿��: %d\n", sDevProp.memoryBusWidth );
		printf( "ʱ��Ƶ��: %d\n", sDevProp.clockRate );
		hipSetDevice(numDevices);
	}


	getchar();
	return 0;
Error:
	fprintf(stderr,"CUDA failure code : 0x%x\n",status);
	return 1;
}


