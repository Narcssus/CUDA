#include "hip/hip_runtime.h"

////////////////////////////////////////////////////////////////////////
// GPU version of Monte Carlo algorithm using NVIDIA's CURAND library
////////////////////////////////////////////////////////////////////////

#include "hip/hip_runtime.h"
#include ""

#include <stdlib.h>
#include <stdio.h>
#include <math.h>

#include "cutil_inline.h"

#include "hip/hip_runtime.h"
#include <hiprand.h>
#include <Windows.h>
////////////////////////////////////////////////////////////////////////
// CUDA global constants
////////////////////////////////////////////////////////////////////////

__constant__ int   N;
__constant__ float T, r, sigma, rho, alpha, dt, con1, con2;


////////////////////////////////////////////////////////////////////////
// kernel routine
////////////////////////////////////////////////////////////////////////


__global__ void pathcalc(float *d_z, float *d_v)
{
  float s1, s2, y1, y2, payoff;

  // move array pointers to correct position

  // version 1
  //d_z = d_z + threadIdx.x + 2*N*blockIdx.x*blockDim.x;

  // version 2
   d_z = d_z + 2*N*threadIdx.x + 2*N*blockIdx.x*blockDim.x;

  d_v = d_v + threadIdx.x +     blockIdx.x*blockDim.x;

  // path calculation

  s1 = 1.0f;
  s2 = 1.0f;

  for (int n=0; n<N; n++) {
    y1   = (*d_z);
    // version 1
    //d_z += blockDim.x;      // shift pointer to next element
    // version 2
    d_z += 1; 

    y2   = rho*y1 + alpha*(*d_z);
    // version 1
     // d_z += blockDim.x;      // shift pointer to next element
    // version 2
     d_z += 1; 

    s1 = s1*(con1 + con2*y1);
    s2 = s2*(con1 + con2*y2);
  }

  // put payoff value into device array

  payoff = 0.0f;
  if ( fabs(s1-1.0f)<0.1f && fabs(s2-1.0f)<0.1f ) payoff = exp(-r*T);

  *d_v = payoff;
}


////////////////////////////////////////////////////////////////////////
// Main program
////////////////////////////////////////////////////////////////////////

int main(int argc, char **argv){
    
  int     NPATH=960000, h_N=100;
  float   h_T, h_r, h_sigma, h_rho, h_alpha, h_dt, h_con1, h_con2;
  float  *h_v, *d_v,*d_z;
  double  sum1, sum2;

  LARGE_INTEGER t_start,t_end,freq;
  float ms;
  QueryPerformanceFrequency(&freq);
  hiprandGenerator_t gen;
  // initialise card

  cutilDeviceInit(argc, argv);

  // allocate memory on host and device

  h_v = (float *)malloc(sizeof(float)*NPATH);

  cudaSafeCall( hipMalloc((void **)&d_v, sizeof(float)*NPATH) );
  cudaSafeCall( hipMalloc((void **)&d_z, sizeof(float)*2*h_N*NPATH) );

  // define constants and transfer to GPU

  h_T     = 1.0f;
  h_r     = 0.05f;
  h_sigma = 0.1f;
  h_rho   = 0.5f;
  h_alpha = sqrt(1.0f-h_rho*h_rho);
  h_dt    = 1.0f/h_N;
  h_con1  = 1.0f + h_r*h_dt;
  h_con2  = sqrt(h_dt)*h_sigma;

  cudaSafeCall( hipMemcpyToSymbol(HIP_SYMBOL(N),    &h_N,    sizeof(h_N)) );
  cudaSafeCall( hipMemcpyToSymbol(HIP_SYMBOL(T),    &h_T,    sizeof(h_T)) );
  cudaSafeCall( hipMemcpyToSymbol(HIP_SYMBOL(r),    &h_r,    sizeof(h_r)) );
  cudaSafeCall( hipMemcpyToSymbol(HIP_SYMBOL(sigma),&h_sigma,sizeof(h_sigma)) );
  cudaSafeCall( hipMemcpyToSymbol(HIP_SYMBOL(rho),  &h_rho,  sizeof(h_rho)) );
  cudaSafeCall( hipMemcpyToSymbol(HIP_SYMBOL(alpha),&h_alpha,sizeof(h_alpha)) );
  cudaSafeCall( hipMemcpyToSymbol(HIP_SYMBOL(dt),   &h_dt,   sizeof(h_dt)) );
  cudaSafeCall( hipMemcpyToSymbol(HIP_SYMBOL(con1), &h_con1, sizeof(h_con1)) );
  cudaSafeCall( hipMemcpyToSymbol(HIP_SYMBOL(con2), &h_con2, sizeof(h_con2)) );

  // random number generation

  
  QueryPerformanceCounter(&t_start);

 
  hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT);
  hiprandSetPseudoRandomGeneratorSeed(gen, 1234ULL);
  hiprandGenerateNormal(gen, d_z, 2*h_N*NPATH, 0.0f, 1.0f);
 
  cudaSafeCall( hipDeviceSynchronize() );


  QueryPerformanceCounter(&t_end);
  ms=1e3*(t_end.QuadPart-t_start.QuadPart)/freq.QuadPart;
  QueryPerformanceCounter(&t_start);

  printf("\nCURAND normal RNG execution time (ms): %f ,   samples/sec: %e \n",
         ms, 2.0*h_N*NPATH/ms);

  // execute kernel and time it

  pathcalc<<<NPATH/64, 64>>>(d_z, d_v);
  cudaCheckMsg("pathcalc execution failed\n");
  cudaSafeCall( hipDeviceSynchronize() );

  QueryPerformanceCounter(&t_end);
  ms=1e3*(t_end.QuadPart-t_start.QuadPart)/freq.QuadPart;
  QueryPerformanceCounter(&t_start);
  

  printf("Monte Carlo kernel execution time (ms): %f \n",ms);

  // copy back results

  cudaSafeCall( hipMemcpy(h_v, d_v, sizeof(float)*NPATH,
                 hipMemcpyDeviceToHost) );

  // compute average

  sum1 = 0.0;
  sum2 = 0.0;
  for (int i=0; i<NPATH; i++) {
    sum1 += h_v[i];
    sum2 += h_v[i]*h_v[i];
  }

  printf("\nAverage value and standard deviation of error  = %13.8f %13.8f\n\n",
	 sum1/NPATH, sqrt((sum2/NPATH - (sum1/NPATH)*(sum1/NPATH))/NPATH) );

  // Tidy up library

  hiprandDestroyGenerator(gen);

  // Release memory and exit cleanly

  free(h_v);
  cudaSafeCall( hipFree(d_v) );
  cudaSafeCall( hipFree(d_z) );

  // CUDA exit -- needed to flush printf write buffer

  hipDeviceReset();
  getchar();
  return 0;
}
