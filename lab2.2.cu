#include "hip/hip_runtime.h"


#include <stdio.h>
#include <math.h>
#include<time.h>
#include<stdlib.h>

#define arrayLength 10000
float a[10240000],b[10240000];
float c[1048576];
hipError_t addWithCuda(float *c, const float *a, const float *b, unsigned int size);

double randf() 
{ 
	return (double)(rand()/(double)RAND_MAX); 
}
__global__ void calculWithGPU(float *c, const float *a, const float *b,int k)
{
	int i = blockIdx.x;
	int j = threadIdx.x;
	c[i+j*1024]=c[i+j*1024]+(a[i*1024+k]-b[j*1024+k])*(a[i*1024+k]-b[j*1024+k]);
}

void calculWithCPU()
{
	for(int i=0;i<1024;i++)
	{
		for(int j=0;j<1024;j++)
		{
			for(int k=0;k<arrayLength;k++)
			{
				c[i+j*1024]=c[i+j*1024]+(a[i*1024+k]-b[j*1024+k])*(a[i*1024+k]-b[j*1024+k]);
			}
			c[i+j*1024]=sqrtf(c[i+j*1024]);
		}
	}

}

int main()
{
	hipEvent_t start = 0;
	hipEvent_t stop = 0;
	time_t t_start,t_end;  
	//*********�������������������**********************
	srand(time(NULL)); 
	for(int i=0;i<1024;i++){
		for(int j=0;j<arrayLength;j++){
			a[i*arrayLength+j]=randf();
		}
	}
	for(int i=0;i<1024;i++){
		for(int j=0;j<arrayLength;j++){
			b[i*arrayLength+j]=randf();
		}
	}
	for(int i=0;i<1048576;i++)
	{
		c[i]=0;
	}
	//*********������CPU����*******************************
	/*
	t_start = time(NULL) ;
	calculWithCPU();
	t_end = time(NULL) ;
	printf("CPU spends %ld s to finish the mission.Press ENTER to see the data\n",t_end-t_start );
	getchar();
	for(int i=0;i<1024*1024;i++)
	{
		printf("%f\n",c[i]);
	}
	getchar();
	*/
	//*************������GPU����****************************
	
	t_start = time(NULL) ;
	hipError_t cudaStatus = addWithCuda(c, a, b, 10240000);
	for(int i=0;i<1024*1024;i++)
	{
		c[i]=sqrtf(c[i]);
	}
	t_end = time(NULL) ;
	printf("GPU spends %ld s to finish the mission.Press ENTER to see the data\n",t_end-t_start );
	getchar();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "addWithCuda failed!");
		return 1;
	}

	for(int i=0;i<1024*1024;i++)
	{
		printf("%f\n",c[i]);
		//getchar();
	}

	getchar();
	return 0;
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t addWithCuda(float *c, const float *a, const float *b, unsigned int size)
{
	float *dev_a = 0;
	float *dev_b = 0;
	float *dev_c = 0;
	hipError_t cudaStatus;

	// Choose which GPU to run on, change this on a multi-GPU system.
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		goto Error;
	}

	// Allocate GPU buffers for three vectors (two input, one output).
	cudaStatus = hipMalloc((void**)&dev_c, 1048576 * sizeof(float));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(float));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(float));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	// Copy input vectors from host memory to GPU buffers.
	cudaStatus = hipMemcpy(dev_a, a, size * sizeof(float), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	cudaStatus = hipMemcpy(dev_b, b, size * sizeof(float), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}
	cudaStatus = hipMemcpy(dev_c, c, 1048576 * sizeof(float), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}
	// Launch a kernel on the GPU with one thread for each element.
	for(int k=0;k<arrayLength;k++)
		calculWithGPU<<<1024, 1024>>>(dev_c, dev_a, dev_b,k);

	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	// cudaDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
		goto Error;
	}

	// Copy output vector from GPU buffer to host memory.
	cudaStatus = hipMemcpy(c, dev_c, 1048576 * sizeof(float), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

Error:
	hipFree(dev_c);
	hipFree(dev_a);
	hipFree(dev_b);

	return cudaStatus;
}
