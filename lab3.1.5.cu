#include "hip/hip_runtime.h"

////////////////////////////////////////////////////////////////////////
// GPU version of Monte Carlo algorithm using NVIDIA's CURAND library
////////////////////////////////////////////////////////////////////////

#include "hip/hip_runtime.h"
#include ""

#include <stdlib.h>
#include <stdio.h>
#include <math.h>

#include "cutil_inline.h"

#include "hip/hip_runtime.h"
#include <hiprand.h>
#include <Windows.h>
////////////////////////////////////////////////////////////////////////
// CUDA global constants
////////////////////////////////////////////////////////////////////////

__constant__ int   N;
__constant__ float a,b,c;



__global__ void pathcalc(float *x, float *part_result)
{
	float s1, s2, x1, x2, payoff;
	x = x + threadIdx.x + 2*N*blockIdx.x*blockDim.x;

	part_result = part_result + threadIdx.x +     blockIdx.x*blockDim.x;

	// path calculation

	s1 = 0.0f;

	for (int n=0; n<N; n++) {
		x1   = (*x);
		x2   = -(*x);
		x += blockDim.x;     
		s1 = s1+a*x1*x1+b*x1+c;
		s1 = s1+a*x2*x2+b*x2+c;
	}
	*part_result = s1/(2*N);
}



int main(){

	int     size=960000, h_N=100;
	float   aa, bb, cc;
	float  *result, *part_result,*x;
	double  sum;

	hiprandGenerator_t gen;

	result = (float *)malloc(sizeof(float)*size);

	cudaSafeCall( hipMalloc((void **)&part_result, sizeof(float)*size) );
	cudaSafeCall( hipMalloc((void **)&x, sizeof(float)*2*h_N*size) );

	aa = 1.0f;
	bb = 5.0f;
	cc = 9.0f;


	cudaSafeCall( hipMemcpyToSymbol(HIP_SYMBOL(N),    &h_N,    sizeof(h_N)) );
	cudaSafeCall( hipMemcpyToSymbol(HIP_SYMBOL(a),    &aa,    sizeof(aa)) );
	cudaSafeCall( hipMemcpyToSymbol(HIP_SYMBOL(b),    &bb,    sizeof(bb)) );
	cudaSafeCall( hipMemcpyToSymbol(HIP_SYMBOL(c),&cc,sizeof(cc)) );


	hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT);
	hiprandSetPseudoRandomGeneratorSeed(gen, 1234ULL);
	hiprandGenerateNormal(gen, x, 2*h_N*size, -1.0f, 1.0f);

	cudaSafeCall( hipDeviceSynchronize() );

	pathcalc<<<size/64, 64>>>(x, part_result);
	cudaCheckMsg("pathcalc execution failed\n");
	cudaSafeCall( hipDeviceSynchronize() );

	cudaSafeCall( hipMemcpy(result, part_result, sizeof(float)*size,
		hipMemcpyDeviceToHost) );


	sum = 0.0;

	for (int i=0; i<size; i++) {
		sum += result[i];
	}

	printf("a=%f,b=%f,c=%f\nAverage value = %f",aa,bb,cc,sum/size);

	hiprandDestroyGenerator(gen);

	free(result);
	cudaSafeCall( hipFree(part_result) );
	cudaSafeCall( hipFree(x) );

	hipDeviceReset();
	getchar();
	return 0;
}
